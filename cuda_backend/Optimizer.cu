#include "Optimizer.h"
#include "OptimizerKernels.h"

Optimizer::Optimizer(
    unsigned int numThreadsPerBlock,
    const float *h_imageExpected,
    const float *h_imageObserved,
    const float *h_pointSpreadFn,
    const float *h_pointSpreadFnFlip,
    int imageRows,
    int imageCols,
    int imageRowPadding,
    int imageColPadding,
    int pointSpreadFnRows,
    int pointSpreadFnCols)
    :
    numThreadsPerBlock(numThreadsPerBlock),
    imageRows(imageRows),
    imageCols(imageCols),
    imageRowPadding(imageRowPadding),
    imageColPadding(imageColPadding),
    pointSpreadFnRows(pointSpreadFnRows),
    pointSpreadFnCols(pointSpreadFnCols)
{
    // allocate resources on device
    hipMallocPitch(
        &(this->d_imageExpected),
        &(this->imagePitch),
        this->imageCols * sizeof(float),
        this->imageRows);
    hipMallocPitch(
        &(this->d_imageObserved),
        &(this->imagePitch),
        this->imageCols * sizeof(float),
        this->imageRows);
    hipMallocPitch(
        &(this->d_imageIntrinsic),
        &(this->imagePitch),
        this->imageCols * sizeof(float),
        this->imageRows);
    hipMallocPitch(
        &(this->d_imageDifferential),
        &(this->imagePitch),
        this->imageCols * sizeof(float),
        this->imageRows);
    hipMallocPitch(
        &(this->d_pointSpreadFn),
        &(this->pointSpreadFnPitch),
        this->pointSpreadFnCols * sizeof(float),
        this->pointSpreadFnRows);
    hipMallocPitch(
        &(this->d_pointSpreadFnFlip),
        &(this->pointSpreadFnPitch),
        this->pointSpreadFnCols * sizeof(float),
        this->pointSpreadFnRows);

    // copy input data
    hipMemcpy2D(
        this->d_imageExpected,
        this->imagePitch,
        h_imageExpected,
        this->imageCols * sizeof(float),
        this->imageCols * sizeof(float),
        this->imageRows,
        hipMemcpyHostToDevice);
    hipMemcpy2D(
        this->d_imageObserved,
        this->imagePitch,
        h_imageObserved,
        this->imageCols * sizeof(float),
        this->imageCols * sizeof(float),
        this->imageRows,
        hipMemcpyHostToDevice);
    hipMemcpy2D(
        this->d_imageIntrinsic,
        this->imagePitch,
        h_imageExpected,
        this->imageCols * sizeof(float),
        this->imageCols * sizeof(float),
        this->imageRows,
        hipMemcpyHostToDevice);
    hipMemcpy2D(
        this->d_pointSpreadFn,
        this->pointSpreadFnPitch,
        h_pointSpreadFn,
        this->pointSpreadFnCols * sizeof(float),
        this->pointSpreadFnCols * sizeof(float),
        this->pointSpreadFnRows,
        hipMemcpyHostToDevice);
    hipMemcpy2D(
        this->d_pointSpreadFnFlip,
        this->pointSpreadFnPitch,
        h_pointSpreadFnFlip,
        this->pointSpreadFnCols * sizeof(float),
        this->pointSpreadFnCols * sizeof(float),
        this->pointSpreadFnRows,
        hipMemcpyHostToDevice);

    // initialize differential image to zeros
    size_t total = size_t(this->imageRows) * size_t(this->imageCols);
    float *zeros = new float[total];
    for (size_t i = 0; i < total; i ++) {
        zeros[i] = 0.0f;
    }
    hipMemcpy2D(
        this->d_imageDifferential,
        this->imagePitch,
        zeros,
        this->imageCols * sizeof(float),
        this->imageCols * sizeof(float),
        this->imageRows,
        hipMemcpyHostToDevice);
    delete [] zeros;
}

Optimizer::~Optimizer()
{
    // free all device resources
    hipFree(this->d_imageExpected);
    hipFree(this->d_imageObserved);
    hipFree(this->d_imageIntrinsic);
    hipFree(this->d_imageDifferential);
    hipFree(this->d_pointSpreadFn);
    hipFree(this->d_pointSpreadFnFlip);
}

void Optimizer::step(double gradientDescentEta, double regularizerLambda)
{
    unsigned int rows = this->imageRows - 2 * this->imageRowPadding;
    unsigned int cols = this->imageCols - 2 * this->imageColPadding;
    dim3 blockDim(this->numThreadsPerBlock, this->numThreadsPerBlock);
    dim3 gridDim(cols / blockDim.x, rows / blockDim.y);

    evalObjectiveFnDerivative<<<gridDim, blockDim>>>(
        this->d_imageExpected,
        this->d_imageObserved,
        this->d_pointSpreadFnFlip,
        this->d_imageDifferential,
        this->imageRowPadding,
        this->imageColPadding,
        this->pointSpreadFnRows,
        this->pointSpreadFnCols,
        this->imagePitch,
        this->pointSpreadFnPitch);

    if (regularizerLambda != 0.0) {
        evalRegularizerDerivative<<<gridDim, blockDim>>>(
            this->d_imageIntrinsic,
            this->d_imageDifferential,
            this->imageRowPadding,
            this->imageColPadding,
            this->imagePitch,
            regularizerLambda);
    } // else don't use regularizer

    updateObserved<<<gridDim, blockDim>>>(
        this->d_imageDifferential,
        this->d_pointSpreadFn,
        this->d_imageObserved,
        this->imageRowPadding,
        this->imageColPadding,
        this->pointSpreadFnRows,
        this->pointSpreadFnCols,
        this->imagePitch,
        this->pointSpreadFnPitch,
        gradientDescentEta);

    updateIntrinsic<<<gridDim, blockDim>>>(
        this->d_imageDifferential,
        this->d_imageIntrinsic,
        this->imageRowPadding,
        this->imageColPadding,
        this->imagePitch,
        gradientDescentEta);

    zeroDifferential<<<gridDim, blockDim>>>(
        this->d_imageDifferential,
        this->imageRowPadding,
        this->imageColPadding,
        this->imagePitch);
}

void Optimizer::getResultFromDevice(float *h_imageIntrinsic) const
{
    hipMemcpy2D(
        h_imageIntrinsic,
        this->imageCols * sizeof(float),
        this->d_imageIntrinsic,
        this->imagePitch,
        this->imageCols * sizeof(float),
        this->imageRows,
        hipMemcpyDeviceToHost);
}
