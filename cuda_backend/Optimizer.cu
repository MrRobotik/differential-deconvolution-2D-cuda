#include "Optimizer.h"
#include "OptimizerKernels.h"

Optimizer::Optimizer(
    const float *h_imageExpected,
    const float *h_imageObserved,
    const float *h_pointSpreadFn,
    const float *h_pointSpreadFnFlip,
    int imageRows,
    int imageCols,
    int pointSpreadFnRows,
    int pointSpreadFnCols)
    :
    imageRows(imageRows),
    imageCols(imageCols),
    imagePaddedRows(imageRows + (pointSpreadFnRows - 1)),
    imagePaddedCols(imageCols + (pointSpreadFnCols - 1)),
    pointSpreadFnRows(pointSpreadFnRows),
    pointSpreadFnCols(pointSpreadFnCols)
{
    // allocate resources on device
    hipMallocPitch(
        &(this->d_imageExpected),
        &(this->imagePaddedPitch),
        this->imagePaddedCols * sizeof(float),
        this->imagePaddedRows);
    hipMallocPitch(
        &(this->d_imageObserved),
        &(this->imagePaddedPitch),
        this->imagePaddedCols * sizeof(float),
        this->imagePaddedRows);
    hipMallocPitch(
        &(this->d_imageIntrinsic),
        &(this->imagePaddedPitch),
        this->imagePaddedCols * sizeof(float),
        this->imagePaddedRows);
    hipMallocPitch(
        &(this->d_imageDifferential),
        &(this->imagePitch),
        this->imageCols * sizeof(float),
        this->imageRows);
    hipMallocPitch(
        &(this->d_pointSpreadFn),
        &(this->pointSpreadFnPitch),
        this->pointSpreadFnCols * sizeof(float),
        this->pointSpreadFnRows);
    hipMallocPitch(
        &(this->d_pointSpreadFnFlip),
        &(this->pointSpreadFnPitch),
        this->pointSpreadFnCols * sizeof(float),
        this->pointSpreadFnRows);

    // copy input data
    hipMemcpy2D(
        this->d_imageExpected,
        this->imagePaddedPitch,
        h_imageExpected,
        this->imagePaddedCols * sizeof(float),
        this->imagePaddedCols * sizeof(float),
        this->imagePaddedRows,
        hipMemcpyHostToDevice);
    hipMemcpy2D(
        this->d_imageObserved,
        this->imagePaddedPitch,
        h_imageObserved,
        this->imagePaddedCols * sizeof(float),
        this->imagePaddedCols * sizeof(float),
        this->imagePaddedRows,
        hipMemcpyHostToDevice);
    hipMemcpy2D(
        this->d_imageIntrinsic,
        this->imagePaddedPitch,
        h_imageExpected,
        this->imagePaddedCols * sizeof(float),
        this->imagePaddedCols * sizeof(float),
        this->imagePaddedRows,
        hipMemcpyHostToDevice);
    hipMemcpy2D(
        this->d_pointSpreadFn,
        this->pointSpreadFnPitch,
        h_pointSpreadFn,
        this->pointSpreadFnCols * sizeof(float),
        this->pointSpreadFnCols * sizeof(float),
        this->pointSpreadFnRows,
        hipMemcpyHostToDevice);
    hipMemcpy2D(
        this->d_pointSpreadFnFlip,
        this->pointSpreadFnPitch,
        h_pointSpreadFnFlip,
        this->pointSpreadFnCols * sizeof(float),
        this->pointSpreadFnCols * sizeof(float),
        this->pointSpreadFnRows,
        hipMemcpyHostToDevice);

    // prepare execution settings
    this->numThreadsPerBlock.x = 32u;
    this->numThreadsPerBlock.y = 32u;
    this->numBlocks.x = this->imageCols / this->numThreadsPerBlock.x;
    this->numBlocks.y = this->imageRows / this->numThreadsPerBlock.y;
}

Optimizer::~Optimizer()
{
    // free all device resources
    hipFree(this->d_imageExpected);
    hipFree(this->d_imageObserved);
    hipFree(this->d_imageIntrinsic);
    hipFree(this->d_imageDifferential);
    hipFree(this->d_pointSpreadFn);
    hipFree(this->d_pointSpreadFnFlip);
}

void Optimizer::step(double optimizerEta)
{
    dim3 nb = this->numBlocks;
    dim3 nt = this->numThreadsPerBlock;

    zeroDifferential<<<nb, nt>>>(
        this->d_imageDifferential,
        this->imagePitch);

    evalObjectiveFnDerivate<<<nb, nt>>>(
        this->d_imageExpected,
        this->d_imageObserved,
        this->d_pointSpreadFnFlip,
        this->d_imageDifferential,
        this->pointSpreadFnRows,
        this->pointSpreadFnCols,
        this->imagePitch,
        this->imagePaddedPitch,
        this->pointSpreadFnPitch);

    updateObserved<<<nb, nt>>>(
        this->d_imageDifferential,
        this->d_pointSpreadFn,
        this->d_imageObserved,
        this->pointSpreadFnRows,
        this->pointSpreadFnCols,
        this->imagePitch,
        this->imagePaddedPitch,
        this->pointSpreadFnPitch,
        optimizerEta);

    updateIntrinsic<<<nb, nt>>>(
        this->d_imageDifferential,
        this->d_imageIntrinsic,
        this->pointSpreadFnRows,
        this->pointSpreadFnCols,
        this->imagePitch,
        this->imagePaddedPitch,
        optimizerEta);
}

void Optimizer::getResultFromDevice(float *h_imageIntrinsic) const
{
    hipMemcpy2D(
        h_imageIntrinsic,
        this->imagePaddedCols * sizeof(float),
        this->d_imageIntrinsic,
        this->imagePaddedPitch,
        this->imagePaddedCols * sizeof(float),
        this->imagePaddedRows,
        hipMemcpyDeviceToHost);
}
